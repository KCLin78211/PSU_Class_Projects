// Compile using nvcc <file> -lcublas -o <output>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

// Define block size for thread allocation
#define NUM_THREADS 32 // 32 is max for N^2 threads: 32*32 = 1024

//======================================================================================================================
//=== Structure definitions
//======================================================================================================================

typedef struct _kernelParams
{
    int block_size;
    int grid_size;
} sKernelParams;

typedef struct _matrixSize // Optional Command-line multiplier for matrix sizes
{
    unsigned int A_height, A_width, B_height, B_width, C_height, C_width;
} MatrixSize;

typedef struct _vSize // Optional Command-line multiplier for matrix sizes
{
    unsigned int len_A, len_B, len_C;
} VectorSize;


void runTest(int argc, char **argv, int devID);

//======================================================================================================================
//=== Structure functions
//======================================================================================================================

/**
 * @brief -  sets values of vector size structure
 *
 * @param vector_size - pointer to vector size struct
 * @param len - length of all vectors
 */
void SetVectorSize(VectorSize *vector_size, unsigned int len)
{
    vector_size->len_A = len;
    vector_size->len_B = len;
    vector_size->len_C = len;

    printf("Vector A(%u), Vector B(%u), Vector (%u)\n",
           vector_size->len_A,
           vector_size->len_B,
           vector_size->len_C);

    if (vector_size->len_A != vector_size->len_B ||
        vector_size->len_B != vector_size->len_C ||
        vector_size->len_C != vector_size->len_A)
    {
        printf("ERROR: Matrix sizes do not match!\n");
        exit(-1);
    }
}

/**
 * @brief -  sets values of matrix size structure
 *
 * @param matrixSize - reference to matrix size struct
 * @param widthA - width of matrix A
 * @param heightA - height of matrix A
 * @param widthB - width of matrix B
 * @param heightB - height of matrix B
 * @param widthC - width of matrix C
 * @param heightC - height of matrix C
 */
void SetMatrixSize(MatrixSize *matrixSize,
                   unsigned int widthA, unsigned int heightA,
                   unsigned int widthB, unsigned int heightB,
                   unsigned int widthC, unsigned int heightC)
{
    matrixSize->A_height = heightA;
    matrixSize->A_width = widthA;
    matrixSize->B_height = heightB;
    matrixSize->B_width = widthB;
    matrixSize->C_height = heightC;
    matrixSize->C_width = widthC;
}


//======================================================================================================================
//=== GPU memory initialization functions
//======================================================================================================================

/**
 * @brief - allocates memory on GPU for vectors A, B, and C then copies the values for vector A and B
 *          from host PC onto the device
 *
 * @param argc - from compiler
 * @param argv - from compiler
 * @param devID - device ID number
 * @param vector_size - reference to vector size structure
 * @param host_vA - pointer to host vector A (with values)
 * @param host_vB - pointer to host vector B (with values)
 * @param dev_A - pointer to vector A device memory reference
 * @param dev_B - pointer to vector B device memory reference
 * @param dev_C - pointer to vector C device memory reference
 */
void VectorInitCUDA(int argc, char **argv, int devID, VectorSize *vector_size, float *host_vA, float *host_vB,
                    float *&dev_A, float *&dev_B, float *&dev_C)
{
    // Assign CUDA variables
    devID = 0;
    hipGetDevice(&devID);
    hipError_t err;

    // Assign size variables
    size_t size_A = vector_size->len_A * sizeof(float);
    size_t size_B = vector_size->len_B * sizeof(float);
    size_t size_C = vector_size->len_C * sizeof(float);

    // Allocate memory on GPU
    err = hipMalloc((void **) &dev_A, size_A);
    if (err != hipSuccess) printf("Allocate vector A: %s\n", hipGetErrorString(err));
    err = hipMalloc((void **) &dev_B, size_B);
    if (err != hipSuccess) printf("Allocate vector B: %s\n", hipGetErrorString(err));
    err = hipMalloc((void **) &dev_C, size_C);
    if (err != hipSuccess) printf("Allocate vector C: %s\n", hipGetErrorString(err));

    // Copy data from host PC to GPU
    err = hipMemcpy(dev_A, host_vA, size_A, hipMemcpyHostToDevice);
    if (err != hipSuccess) printf("Copy vector A to GPU: %s\n", hipGetErrorString(err));
    err = hipMemcpy(dev_B, host_vB, size_B, hipMemcpyHostToDevice);
    if (err != hipSuccess) printf("Copy vector B to GPU: %s\n", hipGetErrorString(err));

}

/**
 * @brief - allocates memory on GPU for matrices A, B, and C then copies the values for matrices A, B and C
 *          from host PC onto the device
 *
 * @param argc - from compiler
 * @param argv - from compiler
 * @param devID - device ID number
 * @param matrixSize - reference to vector size structure
 * @param host_matrixA - pointer to host matrix A (with values)
 * @param host_matrixB - pointer to host matrix B (with values)
 * @param host_matrixC - pointer to host matrix C (with values)
 * @param dev_matrixA - pointer to matrix A device memory reference
 * @param dev_matrixB - pointer to matrix B device memory reference
 * @param dev_matrixC - pointer to matrix C device memory reference
 */
void MatrixInitCUDA(int argc, char **argv, int &devID, MatrixSize *matrixSize,
                    float *host_matrixA, float *host_matrixB, float *host_matrixC,
                    float *&dev_matrixA, float *&dev_matrixB, float *&dev_matrixC)
{
    // Assign CUDA variables
    devID = 0;
    hipGetDevice(&devID);
    hipError_t err;

    // Assign size variables
    size_t matrixA_size = matrixSize->A_height * matrixSize->A_width * sizeof(float);
    size_t matrixB_size = matrixSize->B_height * matrixSize->B_width * sizeof(float);
    size_t matrixC_size = matrixSize->C_height * matrixSize->C_width * sizeof(float);

    // Allocate memory on GPU
    err = hipMalloc((void **) &dev_matrixA, matrixA_size);
    if (err != hipSuccess) printf("Allocate matrix A: %s\n", hipGetErrorString(err));
    err = hipMalloc((void **) &dev_matrixB, matrixB_size);
    if (err != hipSuccess) printf("Allocate matrix B: %s\n", hipGetErrorString(err));
    err = hipMalloc((void **) &dev_matrixC, matrixC_size);
    if (err != hipSuccess) printf("Allocate matrix C: %s\n", hipGetErrorString(err));

    // Copy data from host PC to GPU
    err = hipMemcpy(dev_matrixA, host_matrixA, matrixA_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) printf("Copy matrix A to GPU: %s\n", hipGetErrorString(err));
    err = hipMemcpy(dev_matrixB, host_matrixB, matrixB_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) printf("Copy matrix B to GPU: %s\n", hipGetErrorString(err));
    err = hipMemcpy(dev_matrixC, host_matrixC, matrixC_size, hipMemcpyHostToDevice);
    if (err != hipSuccess) printf("Copy matrix C to GPU: %s\n", hipGetErrorString(err));
}




//======================================================================================================================
//=== CUDA Vector Kernels
//======================================================================================================================
/**
 * @required ALL VECTORS MUST BE THE SAME LENGTH
 * @brief - kernel for GPU computation of a vector addition
 * @param dev_vecA - pointer to device memory for vector A
 * @param dev_vecB - pointer to device memory for vector B
 * @param dev_vecC - pointer to device memory for vector C
 * @param alpha - multiplier for values in vector A
 * @param beta - multiplier for values in vector B
 * @param vecLen - length of all vectors
 */
__global__ void VectorAdditionKernel(float *dev_vecA, float *dev_vecB, float *dev_vecC,
                                     float alpha, float beta, int vecLen)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < vecLen)
    {
        dev_vecC[i] = alpha*dev_vecA[i] + beta*dev_vecB[i];
    }
}

/**
 * @required ALL VECTORS MUST BE THE SAME LENGTH
 * @brief - kernel for GPU computation of a vector hadamard product
 * @param dev_vecA - pointer to device memory for vector A
 * @param dev_vecB - pointer to device memory for vector B
 * @param dev_vecC - pointer to device memory for vector C
 * @param alpha - multiplier for values in vector A
 * @param beta - multiplier for values in vector B
 * @param vecLen - length of all vectors
 */
__global__ void VectorHadamardKernel(float *dev_vecA, float *dev_vecB, float *dev_vecC,
                                     float alpha, float beta, int vecLen)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < vecLen)
    {
        dev_vecC[i] = alpha*dev_vecA[i] * beta*dev_vecB[i];
    }
}

/**
 * @required ALL VECTORS MUST BE THE SAME LENGTH
 *           REMEMBER: Call kernel using: <<<grid, threads, vecLen>>>
 * @brief - kernel for GPU computation of a vector dot product
 * @param dev_vecA - pointer to device memory for vector A
 * @param dev_vecB - pointer to device memory for vector B
 * @param result - pointer to a single float value where the result will be returned
 * @param alpha - multiplier for values in vector A
 * @param beta - multiplier for values in vector B
 * @param vecLen - length of all vectors
 */
__global__ void VectorDotProduct(float *dev_vecA, float *dev_vecB, float *result,
                                  float alpha, float beta, int vecLen)
{
    extern __shared__ float temp[];
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < vecLen)
    {
        temp[i] = alpha*dev_vecA[i] * beta*dev_vecB[i];
    }
    __syncthreads();
    if(threadIdx.x == 0)
    {
        float sum = 0.0;
        for(int j = 0; j < vecLen; j++)
        {
            sum += temp[j];
        }
        result[0] = sum;
    }
}

//======================================================================================================================
//=== CUDA Vector Kernel Drivers
//======================================================================================================================

/**
 * @brief driver function for computing vector operations
 * @param argc - from compiler
 * @param argv - from compiler
 * @param devID - device ID number
 * @param vectorSize - reference to vector size structure
 * @param operation - switch-case value for which matrix operation to perform
 *                    1: Matrix addition
 *                    2: Matrix Hadamard product
 * @param host_vectorA - pointer to host vector A (with values)
 * @param host_vectorB - pointer to host vector B (with values)
 * @param host_vectorC - pointer to host vector C (with values)
 * @param alpha - multiplier for values in vector A
 * @param beta - multiplier for values in vector B
 */
void RunVectorKernel(int argc, char **argv, int &devID, VectorSize *vectorSize, int operation,
                     float *host_vectorA, float *host_vectorB, float *host_vectorC, float alpha, float beta)
{
    // Assign CUDA variables
    hipError_t err;
    dim3 threads(NUM_THREADS, NUM_THREADS);
    int gridX = (int) ceil((float) vectorSize->len_C / (float) threads.x);
    int gridY = (int) ceil((float) vectorSize->len_C / (float) threads.y);
    dim3 grid((unsigned int) gridX, (unsigned int) gridY);

    // Assign computation variables
    float *dev_vectorA = NULL;
    float *dev_vectorB = NULL;
    float *dev_vectorC = NULL;


    size_t vectorC_size = vectorSize->len_C * sizeof(float);

    // Initialize memory on GPU
    VectorInitCUDA(argc, argv, devID, vectorSize, host_vectorA, host_vectorB, dev_vectorA, dev_vectorB, dev_vectorC);

    switch (operation)
    {
        case 1:
        {
            // Compute vector addition
            VectorAdditionKernel<<<grid, threads>>>(dev_vectorA, dev_vectorB, dev_vectorC, alpha, beta,
                    vectorSize->len_C);
            err = hipGetLastError();
            if (err != hipSuccess) printf("Vector Add Computation: %s\n", hipGetErrorString(err));
            break;
        }
        case 2:
        {
            // Compute vector Hadamard Product
            VectorHadamardKernel<<<grid, threads>>>(dev_vectorA, dev_vectorB, dev_vectorC, alpha, beta,
                    vectorSize->len_C);
            err = hipGetLastError();
            if (err != hipSuccess) printf("Hadamard Computation: %s\n", hipGetErrorString(err));
            break;
        }
        case 3:
        {
            // Compute vector dot product
            VectorDotProduct<<<grid, threads, vectorSize->len_C>>>(dev_vectorA, dev_vectorB, dev_vectorC, alpha, beta, vectorSize->len_C);
            err = hipGetLastError();
            if (err != hipSuccess) printf("Vector Dot product Computation: %s\n", hipGetErrorString(err));
            break;
        }

        default:
            printf("ERROR: No vector kernel selected. Operation Aborted");

    }

    // Make sure device is finished
    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Device synchronize: %s\n", hipGetErrorString(err));

    // Copy data from GPU to host PC
    err = hipMemcpy(host_vectorC, dev_vectorC, vectorC_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess)
        printf("Copy vector C to Host: %s\n", hipGetErrorString(err));

    // Free GPU memory
    err = hipFree(dev_vectorA);
    if (err != hipSuccess) printf("Free vector A on GPU: %s\n", hipGetErrorString(err));
    err = hipFree(dev_vectorB);
    if (err != hipSuccess) printf("Free vector B on GPU: %s\n", hipGetErrorString(err));
    err = hipFree(dev_vectorC);
    if (err != hipSuccess) printf("Free vector C on GPU: %s\n", hipGetErrorString(err));
}

//======================================================================================================================
//=== CUDA Matrix Kernels
//======================================================================================================================

/**
 * @required ALL MATRICES MUST BE THE SAME DIMENSIONS
 * @brief - kernel for GPU computation of matrix additions
 * @param dev_matrixA - pointer to device memory for matrix A
 * @param dev_matrixB - pointer to device memory for matrix B
 * @param dev_matrixC - pointer to device memory for matrix C
 * @param alpha - multiplier for values in matrix A
 * @param beta - multiplier for values in matrix B
 * @param matrix_width - width of all matrices
 * @param matrix_height - height of all matrices
 */
__global__ void MatrixAddKernel(float *dev_matrixA, float *dev_matrixB, float *dev_matrixC,
                                float alpha, float beta, int matrix_width, int matrix_height)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col + row * matrix_height;
    if (col < matrix_width && row < matrix_height)
    {
        dev_matrixC[index] = alpha*dev_matrixA[index] + beta*dev_matrixB[index];
    }
}

/**
 * @required ALL MATRICES MUST BE THE SAME DIMENSIONS
 * @brief - kernel for actual GPU computation for the matrix Hadamard product
 * @param dev_matrixA - pointer to device memory for matrix A
 * @param dev_matrixB - pointer to device memory for matrix B
 * @param dev_matrixC - pointer to device memory for matrix C
 * @param alpha - multiplier for values in matrix A
 * @param beta - multiplier for values in matrix B
 * @param matrix_width - width of all matrices
 * @param matrix_height - height of all matrices
 */
__global__ void MatrixHadamardKernel(float *dev_matrixA, float *dev_matrixB, float *dev_matrixC,
                                     float alpha, float beta, int matrix_width, int matrix_height)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int index = col + row * matrix_height;
    if (col < matrix_width && row < matrix_height)
    {
        dev_matrixC[index] = alpha*dev_matrixA[index] * beta*dev_matrixB[index];
    }
}

//======================================================================================================================
//=== CUDA Matrix Kernel Drivers
//======================================================================================================================

/**
 * @brief -  Uses CUBLAS library to perform alpha(A x B) + beta(C) matrix multiplication and addition
 * @param argc - from compiler
 * @param argv - from compiler
 * @param devID - device ID number
 * @param matrixSize - reference to vector size structure
 * @param host_matrixA - pointer to host matrix A (with values)
 * @param host_matrixB - pointer to host matrix B (with values)
 * @param host_matrixC - pointer to host matrix C (with values)
 * @param alpha - value for alpha in CUBLAS function
 * @param beta - value for beta in CUBLAS function
 * @param transposeA - true if A should be transposed
 * @param transposeB - true if B should be transposed
 */

void MatrixMultiplyCUBLAS(int argc, char **argv, int &devID, MatrixSize *matrixSize,
                          float *host_matrixA, float *host_matrixB, float *host_matrixC,
                          float alpha, float beta, bool transposeA, bool transposeB)
{
    // Assign CUDA variables
    devID = 0;
    hipblasHandle_t handle;
    hipError_t err;
    hipGetDevice(&devID);
    hipblasCreate(&handle);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, devID);
    dim3 threads(NUM_THREADS, NUM_THREADS);
    dim3 grid(matrixSize->C_width / threads.x, matrixSize->C_height / threads.y);

    // Assign computation variables
    float *dev_matrixA = NULL, *dev_matrixB = NULL, *dev_matrixC = NULL;
    int m = matrixSize->A_height;
    int n = matrixSize->B_width;
    int k = matrixSize->A_width;
    hipblasOperation_t transA = HIPBLAS_OP_N, transB = HIPBLAS_OP_N;
    if (transposeA) transA = HIPBLAS_OP_T;
    if (transposeB) transB = HIPBLAS_OP_T;
    size_t matrixC_size = matrixSize->C_height * matrixSize->C_width * sizeof(float);

    // Initialize memory on GPU
    MatrixInitCUDA(argc, argv, devID, matrixSize,
                   host_matrixA, host_matrixB, host_matrixC,
                   dev_matrixA, dev_matrixB, dev_matrixC);


    // Perform matrix multiplication
    // SGEMM PARAMS: (handle, transposeA, transposeB, m, n, k, alpha, matrix A, k, matrix B, n, beta, matrix C, n)
    hipblasSgemm(handle, transA, transB, m, n, k, &alpha, dev_matrixA, k,
                dev_matrixB, n, &beta, dev_matrixC, n);
    err = hipGetLastError();
    if (err != hipSuccess) printf("SGEMM: %s\n", hipGetErrorString(err));

    // Make sure device is finished
    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Device synchronize: %s\n", hipGetErrorString(err));

    // Copy data from GPU to host PC
    err = hipMemcpy(host_matrixC, dev_matrixC, matrixC_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) printf("Copy matrix C to Host: %s\n", hipGetErrorString(err));

    // Free GPU memory
    err = hipFree(dev_matrixA);
    if (err != hipSuccess) printf("Free matrix A on GPU: %s\n", hipGetErrorString(err));
    err = hipFree(dev_matrixB);
    if (err != hipSuccess) printf("Free matrix B on GPU: %s\n", hipGetErrorString(err));
    err = hipFree(dev_matrixC);
    if (err != hipSuccess) printf("Free matrix C on GPU: %s\n", hipGetErrorString(err));
}


/**
 * @required ALL MATRICES MUST BE THE SAME DIMENSIONS
 * @brief driver function for computing the matrix operations
 * @param argc - from compiler
 * @param argv - from compiler
 * @param devID - device ID number
 * @param matrixSize - reference to matrix size structure
 * @param operation - switch-case value for which matrix operation to perform
 *                    1: Matrix addition
 *                    2: Matrix Hadamard product
 * @param host_matrixA - pointer to host matrix A (with values)
 * @param host_matrixB - pointer to host matrix B (with values)
 * @param host_matrixC - pointer to host matrix C (with values)
 * @param alpha - multiplier for values in matrix A
 * @param beta - multiplier for values in matrix B
 */
void RunMatrixKernel(int argc, char **argv, int &devID, MatrixSize *matrixSize, int operation,
                     float *host_matrixA, float *host_matrixB, float *host_matrixC, float alpha, float beta)
{
    // Assign CUDA variables
    hipError_t err;
    dim3 threads(NUM_THREADS, NUM_THREADS);
    int gridX = (int) ceil((float) matrixSize->C_width / (float) threads.x);
    int gridY = (int) ceil((float) matrixSize->C_height / (float) threads.y);
    dim3 grid((unsigned int) gridX, (unsigned int) gridY);

    // Assign computation variables
    float *dev_matrixA = NULL, *dev_matrixB = NULL, *dev_matrixC = NULL;
    size_t matrixC_size = matrixSize->C_height * matrixSize->C_width * sizeof(float);

    // Initialize memory on GPU
    MatrixInitCUDA(argc, argv, devID, matrixSize,
                   host_matrixA, host_matrixB, host_matrixC,
                   dev_matrixA, dev_matrixB, dev_matrixC);

    switch (operation)
    {
        case 1:
        {
            // Compute Matrix Addition
            MatrixAddKernel<<<grid, threads>>>(dev_matrixA, dev_matrixB, dev_matrixC, alpha, beta,
                    matrixSize->C_width, matrixSize->C_height);
            err = hipGetLastError();
            if (err != hipSuccess) printf("Matrix Add Computation: %s\n", hipGetErrorString(err));
            break;
        }
        case 2:
        {
            // Compute Hadamard Product
            MatrixHadamardKernel<<<grid, threads>>>(dev_matrixA, dev_matrixB, dev_matrixC, alpha, beta,
                    matrixSize->C_width, matrixSize->C_height);
            err = hipGetLastError();
            if (err != hipSuccess) printf("Hadamard Computation: %s\n", hipGetErrorString(err));
            break;
        }

        default:
            printf("ERROR: No matrix kernel selected. Operation Aborted");

    }

    // Make sure device is finished
    err = hipDeviceSynchronize();
    if (err != hipSuccess) printf("Device synchronize: %s\n", hipGetErrorString(err));

    // Copy data from GPU to host PC
    err = hipMemcpy(host_matrixC, dev_matrixC, matrixC_size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) printf("Copy matrix C to Host: %s\n", hipGetErrorString(err));

    // Free GPU memory
    err = hipFree(dev_matrixA);
    if (err != hipSuccess) printf("Free matrix A on GPU: %s\n", hipGetErrorString(err));
    err = hipFree(dev_matrixB);
    if (err != hipSuccess) printf("Free matrix B on GPU: %s\n", hipGetErrorString(err));
    err = hipFree(dev_matrixC);
    if (err != hipSuccess) printf("Free matrix C on GPU: %s\n", hipGetErrorString(err));

}

//======================================================================================================================
//=== Main Function
//======================================================================================================================

/**
 * @brief computes weight matrices for a shallow neural network
 * @param argc - from compiler
 * @param argv - from compiler
 * @return 0 if success
 */
int main(int argc, char **argv)
{
    // Create memory for Layer 1, Layer 2, Layer 3 vectors
    // float *layer1 = malloc(784*sizeof(floats)))
    // Create memory for Weight 1->2, Weight 2->3 matrices

    // Layer 1 will read from file for input (X) values
    // Layer 2 and 3 will be calculated
    int devID = 0;
    hipGetDevice(&devID);

    return 0;
}


void runTest(int argc, char **argv, int devID)
{
    int N = 10;
    float *host_A, *host_B, *host_C, *host_D;
    float *host_vA, *host_vB, *host_vC, *host_vD, *host_vE;

    MatrixSize *testMatrixSize = (MatrixSize *) calloc(sizeof(MatrixSize), 1);
    size_t calcSize = N * N * sizeof(float);
    host_A = (float *) calloc(calcSize, 1);
    host_B = (float *) calloc(calcSize, 1);
    host_C = (float *) calloc(calcSize, 1);
    host_D = (float *) calloc(calcSize, 1);
    SetMatrixSize(testMatrixSize, N, N, N, N, N, N);


    VectorSize *testVectorSize = (VectorSize *) calloc (sizeof(VectorSize), 1);
    size_t calcSize_V = N * sizeof(float);
    host_vA = (float *) calloc(calcSize_V, 1);
    host_vB = (float *) calloc(calcSize_V, 1);
    host_vC = (float *) calloc(calcSize_V, 1);
    host_vD = (float *) calloc(calcSize_V, 1);
    host_vE = (float *) calloc(calcSize_V, 1);
    SetVectorSize(testVectorSize, N);

    for (int i = 0; i < N * N; i++)
    {
        host_A[i] = (float)i;
        host_B[i] = (float)i;
    }


    for (int i = 0; i < N; i++)
    {
        host_vA[i] = (float)i;
        host_vB[i] = (float)i;
    }


    printf("Matrix A:\n");
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%6.0f ", host_A[i * j]);
        }
        printf("\n");
    }
    printf("\nMatrix B:\n");

    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%6.0f ", host_B[i * j]);
        }
        printf("\n");
    }

    printf("Vector A:\n");
    for (int i = 0; i < N; i++)
    {
        printf("%6.0f ", host_vA[i]);
    }
    printf("\n");

    printf("\nVector B:\n");

    for (int i = 0; i < N; i++)
    {
        printf("%6.0f ", host_vB[i]);
    }
    printf("\n");

    RunMatrixKernel(argc, argv, devID, testMatrixSize, 1, host_A, host_B, host_C, 1.0, 1.0);
    RunMatrixKernel(argc, argv, devID, testMatrixSize, 2, host_A, host_B, host_D, 1.0, 1.0);
    RunVectorKernel(argc, argv, devID, testVectorSize, 1, host_vA, host_vB, host_vC, 1.0, 1.0);
    RunVectorKernel(argc, argv, devID, testVectorSize, 2, host_vA, host_vB, host_vD, 1.0, 1.0);
    RunVectorKernel(argc, argv, devID, testVectorSize, 3, host_vA, host_vB, host_vE, 1.0, 1.0);

    printf("\nMatrix C:\n");
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%6.0f ", host_C[i * j]);
        }
        printf("\n");
    }
    printf("\nMatrix D:\n");
    for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {
            printf("%6.0f ", host_D[i * j]);
        }
        printf("\n");
    }

    printf("Vector C:\n");
    for (int i = 0; i < N; i++)
    {
        printf("%6.0f ", host_vC[i]);
    }
    printf("\n");

    printf("\nVector D:\n");

    for (int i = 0; i < N; i++)
    {
        printf("%6.0f ", host_vD[i]);
    }
    printf("\n");

    printf("\nVector E:\n");

    for (int i = 0; i < N; i++)
    {
        printf("%6.0f ", host_vE[0]);
    }
    printf("\n");
}